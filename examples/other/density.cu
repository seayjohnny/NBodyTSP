#include "hip/hip_runtime.h"
//nvcc SeayJohnnyHW5.cu -o SeayJohnnyHW5 -lglut -lGL -lm

#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include "./arrays.h"
#include "./drawing.h"
#include "./density.h"

#define DIM 1024
#define NODES 5000

float2 *nodes = (float2*)malloc((NODES)*sizeof(float2));
__constant__ float2 c_nodes[NODES];

float *pixels;
float *buffer;

int trigger = 1;

float rnd(float x)
{
    return(x*rand() / RAND_MAX);
}
void display()
{
    glClear(GL_COLOR_BUFFER_BIT);

    int b = 32;
    drawDensity(nodes, NODES, b, 1.0);
    drawPoints(nodes, NODES, 5.0);
    drawGrid(2.0/b, 2.0/b, 1.0);
    glFlush();
}

int main(int argc, char** argv)
{ 
    srand( time(NULL) );
    for(int i = 0; i < NODES; i++)
    {
        nodes[i].x = rnd(2.0) - 1.0;
        nodes[i].y = rnd(2.0) - 1.0;
    }
    
    hipMemcpyToSymbol(HIP_SYMBOL(c_nodes), nodes, sizeof(float2)*NODES);

    pixels = (float*)malloc(DIM*DIM*3*sizeof(float));
    hipMalloc(&buffer, DIM*DIM*3*sizeof(float));
    
    // Initialize OpenGL
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE | GLUT_MULTISAMPLE);
    glutInitWindowSize(DIM, DIM);

   	glutCreateWindow("GPU | Time to render:\t---");
    glutDisplayFunc(display);

    glClearColor(0.0, 0.0, 0.0, 0.1);
    glEnable(GL_MULTISAMPLE_ARB);
    glEnable(GL_POINT_SMOOTH);
    glHint(GL_POINT_SMOOTH_HINT, GL_NICEST);
    glEnable(GL_BLEND);
    glDisable(GL_DEPTH_TEST);

    glutMainLoop();

    return(0);

}